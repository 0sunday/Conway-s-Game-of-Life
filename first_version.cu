#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define M 32
#define KNRM  "\x1B[0m"
#define KRED  "\x1B[31m"
#define KGRN  "\x1B[32m"
#define KYEL  "\x1B[33m"
#define KBLU  "\x1B[34m"
#define KMAG  "\x1B[35m"
#define KCYN  "\x1B[36m"
#define KWHT  "\x1B[37m"


__global__
void uni_func(int *A,int width,int *OUT)
{
		int col = blockIdx.x*blockDim.x + threadIdx.x;
		int row = blockIdx.y*blockDim.y + threadIdx.y;

		int n[8];//neighboors
		//oria kadrou
		bool first_row,last_row,first_col,last_col;
		first_row = col>=0 && col <= width-1;//prwti grammi = 0
		last_row = (col>=(width*width)-width) && (col<=(width*width)-1) ;//teleutaia grammi = 0
		first_col = col%width == 0;//prwti stili = 0
		last_col = col%width == width -1 ;

		if ( first_row || last_row || first_col || last_col ) {
			OUT[col] = 0;
		}
		else{
			n[0] = A[(col-1-width)] ;
			n[1] = A[(col-width)] ;
			n[2] = A[(col+1-width)] ;

			n[3] = A[(col-1)] ;//ditikos geitonas
			int iam = A[row*width + col] ; // κεντρικο κελι
			n[4] = A[(col+1)] ;//anatolikos geitonas

			n[5] = A[(col-1+width)] ;
			n[6] = A[(col+width)] ;
			n[7] = A[(col+1+width)] ;



			//on || off || dying
			//3 counters
			int counter_alive=0;
			int counter_dead=0;
			int counter_DYING=0;

			// rules: -1: dying && 0:off && 1:on
			//to parakatw for einai gia ton ipologismo twn counter

			for (int i = 0; i <= 7; i++)
			{
				if (n[i] != -1)//for sure is not dying - actually is not -1(negative number)
				{
					counter_alive += n[i];//counter_alive = counter_alive + 0/1
				}
				else//n[i] = -1
				{
					counter_DYING -= n[i] ;//-(-1)=+1
				}
			}
			counter_dead = 8 - ( counter_alive + counter_DYING);//all neighboors - not_dead


			//ti na kanw me emena???
			//parakatw tha tsekaroume to thread kai tous geitones tou
			if(iam == -1)//i am dying
			{
				iam = 0;//i will be off
			}
			else if(iam == 1)//i am on
			{
			 	iam = -1;	//i am dying
			}
			else if(iam == 0 && counter_alive == 2 )//i am off and 2 neighboors on
			{
				iam = 1;	//i will be on
			}

			//update me
			OUT[row*width + col] = iam;
		}//end else

}

int main() {

	int i,j;
	int on=0;
	int off=0;
	int dying=0;
	int N=M*M;//all elements of A
	int A[M][M] ;
	int OUT[M][M] ;
	srand (time(NULL));
	printf("\n....IN MAIN...\n");
	for(i=0;i< M;i++)
	{
		for(j=0;j< M;j++)
		{
			if (i==0 || i==M-1 || j==M-1 || j==0){
				A[i][j] = 0;//to perigramma tou pinaka
				OUT[i][j] = 0;
			}
			else{
				A[i][j]=  rand()%3 -1;

				OUT[i][j] = -999;
			}
		}
		//printf("\n");
	}
	for(i=0;i< M;i++)
	{
		for(j=0;j< M;j++)
		{
			if (A[i][j] == -1){printf("%d ", A[i][j]);}
			else{printf(" %d ", A[i][j]);}
		}
		printf("\n");
	}
	//launching kernel

	int *A_device;//int A_size = N*sizeof(int) ;
	const size_t A_size = sizeof(int) * size_t(N);
	hipMalloc((void **)&A_device, A_size);

	int *OUT_device;//int A_size = N*sizeof(int) ;
	const size_t OUT_size = sizeof(int) * size_t(N);
	hipMalloc((void **)&OUT_device, OUT_size);

	hipMemcpy(A_device, A, A_size, hipMemcpyHostToDevice);
	hipMemcpy(OUT_device, OUT, OUT_size, hipMemcpyHostToDevice);


	//the game is on Mrs. Hudson :)

	int turn = 0;

	while (1){

		if (turn % 2 == 0){//zigos arithmos seiras: A->in, Out->Out
			/*
			//VGALE ME AN THES NA DEIS XRONO EKTELESI
			//THIS_BLOCK_IN
			hipEvent_t start,stop;
			float elapsedTime;
			hipEventCreate(&start);
			hipEventRecord(start,0);
			uni_func<<<M,M>>>(A_device,M,OUT_device);
			hipEventCreate(&stop);
			hipEventRecord(stop,0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsedTime,start,stop);
			printf("\net:%f\n",elapsedTime);
			break;
			//END_OF_BLOCK
			*/

			//VGALE TIN KATW AN THES NA DEIS XRONO EKTELESI
			uni_func<<<M,M>>>(A_device,M,OUT_device);
			hipMemcpy(OUT, OUT_device, A_size,  hipMemcpyDeviceToHost);//thats work
			printf("\n\n-------------\n\n%d Time\n\n\n\n",turn);

			for(i=0;i< M;i++)
			{
				for(j=0;j< M;j++)
				{
					if (OUT[i][j] == -1){printf("%s%d ",KRED, OUT[i][j]);}
					else if (OUT[i][j] == 1){printf(" %s%d ",KGRN, OUT[i][j]);}
					else{printf(" %s%d ",KNRM, OUT[i][j]);}

					//make counter
					if (OUT[i][j] == -1){ dying++;}
					else if (OUT[i][j] == 1) {on++;}
					else {off++;}


				}
				printf("\n");
			}
		}
		else{
			uni_func<<<M,M>>>(OUT_device,M,A_device);
			hipMemcpy(A, A_device, A_size,  hipMemcpyDeviceToHost);
			printf("\n\n-------------\n\n%d Time\n\n\n\n",turn);

			for(i=0;i< M;i++)
			{
				for(j=0;j< M;j++)
				{
					if (A[i][j] == -1){printf("%s%d ",KRED, A[i][j]);}
					else if (A[i][j]==1){printf(" %s%d ",KGRN, A[i][j]);}
					else {printf(" %s%d ",KNRM, A[i][j]);}

					//make counter
					if (A[i][j] == -1){ dying++;}
					else if (A[i][j] == 1) {on++;}
					else {off++;}
				}
				printf("\n");
			}
		}
		//print counter
		printf("\n%s----------------------------------------------------\n",KNRM);
		printf("counter_alive: %d, counter_dying: %d, counter_dead: %d\n",on,dying,off);
		printf("--------------------------------------------------------\n");
		//counters = 0
		if (off == N){break;}//all elements are off (N=M*M)
		on = 0;
		off = 0;
		dying = 0;
		turn++;//auksanoume seira gia na kalesoume uni_func me allagi eisodwn-eksodwn


	}



	return 0;
}
