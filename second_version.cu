#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define M 32
#define KNRM  "\x1B[0m"
#define KRED  "\x1B[31m"
#define KGRN  "\x1B[32m"
#define KYEL  "\x1B[33m"
#define KBLU  "\x1B[34m"
#define KMAG  "\x1B[35m"
#define KCYN  "\x1B[36m"
#define KWHT  "\x1B[37m"


__global__
void uni_func(int *A,int width,int *OUT)
{
//	OUT[33] = 44444;

	__shared__ int ns[M*M];//neighboors state
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	//limits
	/* mind blowing	*/
	bool first_row,last_row,first_col,last_col;
	first_row = col>=0 && col <= width-1;//first row
	last_row = (col>=(width*width)-width) && (col<=(width*width)-1) ;//last row
	first_col = col%width == 0;//first column
	last_col = col%width == width -1 ;//last column


	if ((first_row || last_row)){OUT[col]=0;}
	else{// to thread den einai sto "orizontio" kadro
		/*osa stoixeia den anikoun sto "orizontio kadro" tha feroun
		ton panw geitona tous kai ton katw!
		Auta pou anikoun sto "orizontio" kadro an prospathisoun na feroun ton panw i katw
		tha vgoun ektos oriwn kai tha dimiourgisoun provlima */


		ns[col] = A[col];//i katastasi tou thread

		ns[col-width]= A[col-width];//o panw geitonas tou
		ns[col+width] =A[col+width];//o katw geitonas tou
		__syncthreads();//perimenoume ola ta threads na feroun auta pou prepei

		if ((last_col || first_col)){OUT[col]=0;}
		else{//to thread den einai sto "katheto" kadro
		/*mono ta stoixeia pou den einai se kadro tha elenksoun geitones kai
		tha enimerwsoun ton eauto tous sto pinaka OUT.
		Auto ginetai gia 2 logous:
		 1)Osa einai se kadro mporei na vgoun ektos oriwn
		tou pinaka ns (stin shared mem)
		2)etsi kai alliws tha parameinoun 0
		*/

							int n[8];//oi 8 geitones
							n[0] = ns[(col-1-width)] ;//voreio-ditikos
							n[1] = ns[(col-width)] ;//voreios
							n[2] = ns[(col+1-width)] ;//voreio-anatolikos

							n[3] = ns[(col-1)] ;//ditikos
							int iam = ns[col] ; // this is me
							n[4] = ns[(col+1)] ;//anatolikos

							n[5] = ns[(col-1+width)] ;//noteio-ditikos
							n[6] = ns[(col+width)] ;//notios
							n[7] = ns[(col+1+width)] ;//notio-anatolikos



							int counter_alive=0;//on	(1)
							int counter_dead=0;	//off	(0)
							int counter_DYING=0;//dying (-1)

							// rules: -1: dying & 0:off & 1:on

							//elegxos katastasewn geitonwn
							for (int i = 0; i <= 7; i++)
							{
								if (n[i] != -1)//for sure: is not dying - actually is not -1(negative number)
								{
									counter_alive += n[i];//counter_alive = counter_alive + 0/1
								}
								else//
								{
									counter_DYING -= n[i] ;// -(-1)=+1
								}
							}
							counter_dead = 8 - ( counter_alive + counter_DYING);//all neighboors - not_dying


							//ti eimai egw??
							if(iam == -1)//i am dying
							{
								iam = 0;//i will be off
							}
							else if(iam == 1)//i am on
							{
								iam = -1;	//i will  die
							}
							else if(iam == 0 && counter_alive == 2 )//i am off and 2 neighboors on
							{
								iam = 1;	//i will be on

							}

							//update me
							OUT[col] = iam;


			}

	}

}

int main() {
	int i,j;
	int on=0;
	int off=0;
	int dying=0;
	int N=M*M;//all elements of A
	int A[M][M] ;
	int OUT[M][M] ;
	srand (time(NULL));
	printf("\n....IN MAIN...\n");
	//make A
	for(i=0;i< M;i++)
	{
		for(j=0;j< M;j++)
		{
			if (i==0 || i==M-1 || j==M-1 || j==0){
				A[i][j] = 0;//to perigramma tou pinaka
				OUT[i][j] = 0;
			}
			else{
				A[i][j]=  rand()%3 -1;
				OUT[i][j] = -9;
			}
		}
	}

	//print A
	for(i=0;i< M;i++)
	{
		for(j=0;j< M;j++)
		{
			if (A[i][j] == -1){printf("%d ", A[i][j]);}
			else{printf(" %d ", A[i][j]);}
		}
		printf("\n");
	}

	//launching kernel

	int *A_device;	//int A_size = N*sizeof(int) ;
	const size_t A_size = sizeof(int) * size_t(N);
	hipMalloc((void **)&A_device, A_size);

	int *OUT_device;//int A_size = N*sizeof(int) ;
	const size_t OUT_size = sizeof(int) * size_t(N);
	hipMalloc((void **)&OUT_device, OUT_size);

	hipMemcpy(A_device, A, A_size, hipMemcpyHostToDevice);
	hipMemcpy(OUT_device, OUT, OUT_size, hipMemcpyHostToDevice);


	//the game is on Mrs. Hudson :)

	int turn = 0;

	while (1){

		if (turn % 2 == 0){//zigos arithmos seiras: A->in, Out->Out
			/* //VGALE ME AN THES NA DEIS XRONO EKTELESI
			//THIS_BLOCK_IN
			hipEvent_t start,stop;
			float elapsedTime;
			hipEventCreate(&start);
			hipEventRecord(start,0);
			uni_func<<<M,M>>>(A_device,M,OUT_device);
			hipEventCreate(&stop);
			hipEventRecord(stop,0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsedTime,start,stop);
			printf("\net:%f\n",elapsedTime);
			break;
			//END_OF_BLOCK
			*/

			//VGALE TIN KATW AN THES NA DEIS XRONO EKTELESI
			uni_func<<<M,M>>>(A_device,M,OUT_device);
			hipMemcpy(OUT, OUT_device, A_size,  hipMemcpyDeviceToHost);//thats work
			printf("\n\n-------------\n\n%d Time\n\n\n\n",turn);

			for(i=0;i< M;i++)
			{
				for(j=0;j< M;j++)
				{
					if (OUT[i][j] == -1){printf("%s%d ",KRED, OUT[i][j]);}
					else if (OUT[i][j] == 1){printf(" %s%d ",KGRN, OUT[i][j]);}
					else{printf(" %s%d ",KNRM, OUT[i][j]);}

					//make counter
					if (OUT[i][j] == -1){ dying++;}
					else if (OUT[i][j] == 1) {on++;}
					else {off++;}


				}
				printf("\n");
			}
		}
		else{
			uni_func<<<M,M>>>(OUT_device,M,A_device);
			hipMemcpy(A, A_device, A_size,  hipMemcpyDeviceToHost);
			printf("\n\n-------------\n\n%d Time\n\n\n\n",turn);

			for(i=0;i< M;i++)
			{
				for(j=0;j< M;j++)
				{
					if (A[i][j] == -1){printf("%s%d ",KRED, A[i][j]);}
					else if (A[i][j]==1){printf(" %s%d ",KGRN, A[i][j]);}
					else {printf(" %s%d ",KNRM, A[i][j]);}

					//make counter
					if (A[i][j] == -1){ dying++;}
					else if (A[i][j] == 1) {on++;}
					else {off++;}
				}
				printf("\n");
			}
		}
		//print counter
		printf("\n%s----------------------------------------------------\n",KNRM);
		printf("counter_alive: %d, counter_dying: %d, counter_dead: %d\n",on,dying,off);
		printf("--------------------------------------------------------\n");
		//counters = 0
		if (off == N){break;}//all elements are off (N=M*M)
		on = 0;
		off = 0;
		dying = 0;
		turn++;//auksanoume seira gia na kalesoume uni_func me allagi eisodwn-eksodwn


	}



	return 0;
}
